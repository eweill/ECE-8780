#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <math.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include "headers/image.h"

#define TILE_WIDTH 16

#define emax(x, y) ((x) >= (y)) ? (x) : (y)
#define emin(x, y) ((x) <= (y)) ? (x) : (y)

__global__ void imgDarken(int *outputImage, int *inputImage,
			int width, int height)
{
	// Create indexing variables (need 2 dimensional)
	int x = ????;
	int y = ????;

	if (x < width && y < height)
	{
		// Calculate image offset
		int offset = ????;
		outputImage[offset] = emax(inputImage[offset] - 75, 0);
	}
}

int main (int argc, char **argv)
{
	int imageWidth, imageHeight;
	int *hostInputImage, *hostOutputImage;
	int *deviceInputImage, *deviceOutputImage;

	if (argc != 3)
	{
		printf("Usage: ./imageDarken <input-image> <output-image-name>\n");
		exit(1);
	}

	// Read in image and convert to readable format
	read_image_template<int>(argv[1], &hostInputImage, &imageWidth, &imageHeight);

	// Set image size information
	int img_size = imageWidth * imageHeight * sizeof(int);

	// Allocate memory for host image (output)
	hostOutputImage = ????;

	// Allocate memory for images (input and output) on GPU
	????
	
	// Copy host input image to device input image
	????

	// Set kernel dimensions and call kernel
	dim3 dimGrid(????,????);
	dim3 dimBlock(????, ????, 1);
	imgDarken <<< dimGrid, dimBlock >>>(deviceOutputImage, deviceInputImage, imageWidth, imageHeight);

	// Copy resulting image back to host
	????

	// Write image to output file	
	write_image_template<int>(argv[2], hostOutputImage, imageWidth, imageHeight);

	// Free device memory (for input and output image)
	????
	
	// Free host memory (for output image)
	????

	return 0;
}
