#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <time.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

// CUDA kernel, Each thread takes care of one element of c
__global__ void vecAdd(double *a, double *b, double *c, int n)
{
	// Get global thread ID
	int idx = ????;

	// Make sure not to go out of bounds
	if (idx < n)
		????
}

int main (int argc, char **argv)
{
	// Size of vectors for addition
	int n = 10000000;

	// Host vectors
	double *h_a, *h_b, *h_c, *h_h;

	// Device vectors
	double *d_a, *d_b, *d_c;

	// Size in bytes for each vector
	size_t bytes = n*sizeof(double);

	// Allocate memory for each vector on host (h_a(b)(c)(h))
	????

	// Allocate memory for each vector on GPU (d_a(b)(c))
	????

	// Initialize all vectors on host
	int i;
	for (i = 0 ; i < n ; i++)
	{
		h_a[i] = sin(i) * sin(i);
		h_b[i] = cos(i) * cos(i);
	}

	// Copy host vectors to device
	????
		
	int blockSize, gridSize;
	// Number of threads in each thread block, number of thread blocks in grid
	blockSize = 1024;
	gridSize = (int)ceil((float)n/blockSize);

	// Execute on CPU and GPU
	clock_t cpu_start, cpu_end;
	clock_t gpu_start, gpu_end;
	cpu_start = clock();
	for (i = 0 ; i < n ; i++)
	{
		h_h[i] = h_a[i] + h_b[i];
	}
	cpu_end = clock();
	gpu_start = clock();

	// Kernel Call
	????

	gpu_end = clock();

	// Copy array back to host
	????

	// Sum up vector c and print result divided by n
	double sum = 0;
	for ( i = 0 ; i < n ; i++ )
		sum += h_c[i];
	printf("final:result: %f\n", sum/n);

	double cpu_time = ((double)cpu_end - cpu_start)/CLOCKS_PER_SEC;
	double gpu_time = ((double)gpu_end - gpu_start)/CLOCKS_PER_SEC;

	printf("CPU Time: %f\nGPU Time: %f\n", cpu_time, gpu_time);

	// Release device memory
	????

	// Release host memory
	????

	return 0;
}
